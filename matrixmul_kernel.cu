#include "hip/hip_runtime.h"
/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"
#define TILE_WIDTH 16


// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
    __shared__ float M_s[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_s[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    
    // Identify the row and column of the P_d element to work on
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;
    float Pvalue = 0;
  
    // Loop over the M_ and N_ tiles to compute the P_d element
    for (int m = 0; m < ceilf(M.width/(float)TILE_WIDTH); ++m) {
       
        // Collaborative loading of M_d and N_d tiles into shared memory
        if(m*blockDim.x + tx < M.width){
		    M_s[ty][tx] = M.elements[Row * M.width + m*TILE_WIDTH+tx];
        }
	    if(m*blockDim.y + ty < N.height) {
	    	N_s[ty][tx] = N.elements[(m*blockDim.y+ty)*N.width+Col];
        }
	__syncthreads();
        
        for (int k = 0; k < blockDim.y; ++k){
            if(m*blockDim.x+k < M.width){
                Pvalue += M_s[ty][k] * N_s[k][tx];
            }
        }

        __syncthreads();   
    }    
    if(Row < P.height && Col < P.width) {
        P.elements[Row*P.width+Col] = Pvalue;
    }

}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
